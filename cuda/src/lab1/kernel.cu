#include "hip/hip_runtime.h"
#include "../../include/helpme.h"


// Je suis un kerne qui multiplie les elements d'une liste par une scalaire pis qui les mets dans une autre liste
__global__ static void Kernel_ScalairArray_Int(int *ArrayA, int k, int *ArrayR,int size) {
    int index =  blockDim.x;
    if (index < size)
        ArrayR[index] = ArrayA[index] * k;
}

int BLOCK_SIZE = 1;
int THREAD_COUNT = 0;


extern "C" hipError_t StartKernel_ScalairArray_Int(int *pArrayA, int k, int *pArrayR, int size) {
    // Assure qu'on peut belle et bien utiliser cuda
    hipError_t cudaStatus = hipSetDevice(0);
    if(cudaStatus != hipSuccess) {
        return cudaStatus;
    }
    //Crée nos pointeur utiliser par cuda
    int *ArrayA, *ArrayR;
    // Alloue l'espace mémoire des deux ArrayRices sur le gpu
    // calcul de l'espace de notre array de pixel qui représente l'image
    size_t memSize = size * sizeof(int);
    cudaStatus = hipMalloc( (void**)&ArrayA,memSize);
    if(cudaStatus != hipSuccess){
        return cudaStatus;
    }
    cudaStatus = hipMalloc( (void**)&ArrayR,memSize);
    if(cudaStatus != hipSuccess) {
        return cudaStatus;
    }
    // Copie l'array de donnnée vers le gpu
    cudaStatus = hipMemcpy(ArrayA,pArrayA, memSize, hipMemcpyHostToDevice);
    if(cudaStatus != hipSuccess) {
        return cudaStatus;
    }

    // Démarre le kernel
    Kernel_ScalairArray_Int<<<size,1>>>((int*)ArrayA,(int)k,(int*)ArrayR,(int)size);

    if(hipDeviceSynchronize() == hipSuccess) {
        printf("Finit d'execution du kernel");
    }

    // Fait une copie de l'array de resultat du gpu vers le cpu
    cudaStatus = hipMemcpy(pArrayR,ArrayR,memSize,hipMemcpyHostToDevice);
    if(cudaStatus != hipSuccess) {
        return cudaStatus;
    }
    hipFree(ArrayA);
    hipFree(ArrayR);


    return hipSuccess;

}

#include <stdio.h>
#include "hip/hip_runtime.h"

typedef unsigned char uchar;

static void HandleError(hipError_t err, const char *file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		return;
		//exit(EXIT_FAILURE);
	}
}

#define HANDLE_ERROR(err) (HandleError(err,__FILE__.__LINE__))

#define HANDLE_NULL(a) { \
    if (a == NULL) { printf("Host memory failed in %s at line %d\n", __FILE__,__LINE__);\
    exit(EXIT_FAILURE);}}

// Round le resultat de a / b a l'int superieur le plus pres
int iDivUp(int a, int b) {
	return (a % b != 0) ? (a / b + 1) : (a / b);
}


// Je suis un kerne qui multiplie les elements d'une liste par une scalaire pis qui les mets dans une autre liste
__global__ static void Kernel_ScalairArray_Int(uchar *ArrayA, int k, uchar *ArrayR,int size) {
    int index =  blockIdx.x * blockDim.x + threadIdx.x;
    ArrayR[index] = ArrayA[index] * k;
}

int BLOCK_SIZE = 50;

extern "C" hipError_t StartKernel_ScalairArray_Int(uchar *pArrayA, int k, uchar *pArrayR, int size) {

	// Calcul le nombre de thread par bloc que j'ai besoin
    int BLOCK_COUNT = iDivUp(size,BLOCK_SIZE);

    printf("Starting cuda kernel with %d 1D Blocks and %d 1D Threads\r\n",BLOCK_COUNT,BLOCK_SIZE);


    // Assure qu'on peut belle et bien utiliser cuda
    hipError_t cudaStatus = hipSetDevice(0);
    if(cudaStatus != hipSuccess) {
        return cudaStatus;
    }
    //Crée nos pointeur utiliser par cuda
    uchar *ArrayA, *ArrayR;

    // Alloue l'espace mémoire des deux ArrayRices sur le gpu
    // calcul de l'espace de notre array de pixel qui représente l'image
    size_t memSize = size * sizeof(uchar);
    cudaStatus = hipMalloc( (void**)&ArrayA,memSize);
    
	if(cudaStatus != hipSuccess){
        return cudaStatus;
    }
    
	cudaStatus = hipMalloc( (void**)&ArrayR,memSize);
    if(cudaStatus != hipSuccess) {
        return cudaStatus;
    }
    
	// Copie l'array de donnnée vers le gpu
    cudaStatus = hipMemcpy(ArrayA,pArrayA, memSize, hipMemcpyHostToDevice);
    if(cudaStatus != hipSuccess) {
        return cudaStatus;
    }

    // Démarre le kernel
    Kernel_ScalairArray_Int<<<BLOCK_COUNT,BLOCK_SIZE>>>((uchar*)ArrayA,(int)k,(uchar*)ArrayR,(int)size);

    if(hipDeviceSynchronize() == hipSuccess) {
        printf("Finit d'execution du kernel\r\n");
    }

    // Fait une copie de l'array de resultat du gpu vers le cpu
    cudaStatus = hipMemcpy(pArrayR,ArrayR,memSize,hipMemcpyDeviceToHost);
    if(cudaStatus != hipSuccess) {
        return cudaStatus;
    }
    hipFree(ArrayA);
    hipFree(ArrayR);


    return hipSuccess;

}
